#include "hip/hip_runtime.h"
/* -*- C -*- */

#include "clipAreaKernel.h"

#include <assert.h>
#include <stdio.h>

#if 0
__device__
bool
isInside(const Ball ball, const float3 p) {
   return (ball.center.x - p.x) * (ball.center.x - p.x) +
      (ball.center.y - p.y) * (ball.center.y - p.y) +
      (ball.center.z - p.z) * (ball.center.z - p.z) < ball.squaredRadius;
}
#endif

__device__
bool
isInside(const float3 center, const float3 p) {
   return (center.x - p.x) * (center.x - p.x) +
      (center.y - p.y) * (center.y - p.y) +
      (center.z - p.z) * (center.z - p.z) < 1;
}

// Calculate the number of active points after clipping.
__global__
void
clip(const float* referenceMesh, const unsigned meshSize,
     const float3* centers, unsigned centersSize,
     const unsigned* delimiters, const float3* clippingCenters,
     unsigned* activeCounts) {
   // The n_th thread block works on the n_th ball. The following three 
   // variables are the same for all threads in the block.
   __shared__ float3 center;
   __shared__ unsigned begin;
   __shared__ unsigned end;
   if (threadIdx.x == 0) {
      center = centers[blockIdx.x];
      begin = delimiters[blockIdx.x];
      end = delimiters[blockIdx.x + 1];
   }
   __syncthreads();
   // Count the clipped points in parallel.
   __shared__ unsigned clippedCounts[ThreadsPerBlock];
   clippedCounts[threadIdx.x] = 0;
   float3 p;
   bool clipped;
   if (end - begin <= ThreadsPerBlock) {
      // Copy the clipping centers into shared memory.
      __shared__ float3 cc[ThreadsPerBlock];
      const unsigned size = end - begin;
      if (threadIdx.x < size) {
         cc[threadIdx.x] = clippingCenters[begin + threadIdx.x];
      }
      __syncthreads();
      for (unsigned i = threadIdx.x; i < meshSize; i += blockDim.x) {
         // Translate the reference mesh point to lie on the ball.
         p.x = referenceMesh[i] + center.x;
         p.y = referenceMesh[i+meshSize] + center.y;
         p.z = referenceMesh[i+2*meshSize] + center.z;
         // Determine if the point is clipped.
         clipped = false;
         for (unsigned j = 0; j != size; ++j) {
            // Note: Using the bitwise or is more efficient than using the
            // logical or. The latter causes divergent branches.
            clipped |= isInside(cc[j], p);
         }
         clippedCounts[threadIdx.x] += clipped;
      }
   }
   else {
      for (unsigned i = threadIdx.x; i < meshSize; i += blockDim.x) {
         // Translate the reference mesh point to lie on the ball.
         p.x = referenceMesh[i] + center.x;
         p.y = referenceMesh[i+meshSize] + center.y;
         p.z = referenceMesh[i+2*meshSize] + center.z;
         // Determine if the point is clipped.
         clipped = false;
         for (unsigned j = begin; j != end; ++j) {
            // Note: Using the bitwise or is more efficient than using the
            // logical or. The latter causes divergent branches.
            clipped |= isInside(clippingCenters[j], p);
         }
         clippedCounts[threadIdx.x] += clipped;
      }
   }
   // Calculate the total number of clipped points in the first thread.
   __syncthreads();
   if (threadIdx.x == 0) {
      unsigned count = 0;
      for (unsigned i = 0; i != ThreadsPerBlock; ++i) {
         count += clippedCounts[i];
      }
      activeCounts[blockIdx.x] = meshSize - count;
   }
}

extern "C"
void
clipKernel(const float* referenceMesh, const unsigned meshSize, 
           const float3* centers, unsigned centersSize, 
           const unsigned* delimiters, const float3* clippingCenters,
           unsigned* activeCounts) {
   // The mesh size must be a multiple of the number of threads per block.
   assert(meshSize % ThreadsPerBlock == 0);
   // We choose the number of blocks to be the number of balls. The maximum
   // number of blocks is 65535.
   assert(centersSize < (1 << 16 - 1));
   // Invoke the kernel.
   clip<<<centersSize, ThreadsPerBlock>>>
      (referenceMesh, meshSize, centers, centersSize, delimiters,
       clippingCenters, activeCounts);
}
