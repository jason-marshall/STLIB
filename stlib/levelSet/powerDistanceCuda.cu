#include "hip/hip_runtime.h"
/* -*- C++ -*- */

#include "stlib/levelSet/powerDistanceCuda.h"
#include "stlib/cuda/check.h"
#include "stlib/levelSet/hip/hip_runtime.h"
#ifndef __CUDA_ARCH__
#include "stlib/levelSet/gridCuda.h"
#include "stlib/levelSet/ballsCuda.h"
#endif

#include <cassert>

namespace stlib
{
namespace levelSet {


__global__
void
powerDistanceKernel(float* patch,
                    const float3 lowerCorner,
                    const float spacing,
                    const unsigned numBalls,
                    const float4* ballsSquared) {
   // Calculate the Cartesian location of the grid point.
   const float3 p = {lowerCorner.x + spacing * threadIdx.x,
                     lowerCorner.y + spacing * threadIdx.y,
                     lowerCorner.z + spacing * threadIdx.z};
   // Initialize the grid value to infinity.
   float g = 1./0;
   // Find the minimum power distance over the set of balls.
   float d;
   float4 b;
   for (unsigned i = 0; i != numBalls; ++i) {
      b = ballsSquared[i];
      d = (p.x - b.x) * (p.x - b.x) + (p.y - b.y) * (p.y - b.y) +
         (p.z - b.z) * (p.z - b.z) - b.w;
      if (d < g) {
         g = d;
      }
   }
   // Record the patch value. Convert the multi-index to a single offset.
   patch[threadIdx.x + threadIdx.y * blockDim.x +
         threadIdx.z * blockDim.x * blockDim.y] = g;
}


#ifndef __CUDA_ARCH__
// Translate the data to CUDA format and call powerDistanceKernel().
void
powerDistanceCuda(container::EquilateralArray<float, 3, PatchExtent>* patch,
                  const std::array<float, 3>& lowerCorner,
                  const float spacing,
                  const std::vector<geom::BallSquared<float, 3> >& balls) {
   // Allocate memory for the patch.
   float* patchDev;
   CUDA_CHECK(hipMalloc((void**)&patchDev, patch->size() * sizeof(float)));

   // Allocate memory for the balls and copy the memory.
   float4* ballsDev;
   CUDA_CHECK(hipMalloc((void**)&ballsDev, balls.size() * sizeof(float4)));
   {
      std::vector<float4> buffer(balls.size());
      for (std::size_t i = 0; i != balls.size(); ++i) {
         buffer[i].x = balls[i].center[0];
         buffer[i].y = balls[i].center[1];
         buffer[i].z = balls[i].center[2];
         buffer[i].w = balls[i].squaredRadius;
      }
      CUDA_CHECK(hipMemcpy(ballsDev, &buffer[0],
                            buffer.size() * sizeof(float4),
                            hipMemcpyHostToDevice));
   }

   const dim3 ThreadsPerBlock(PatchExtent, PatchExtent, PatchExtent);
   const float3 lower = {lowerCorner[0], lowerCorner[1], lowerCorner[2]};
   powerDistanceKernel<<<1,ThreadsPerBlock>>>
      (patchDev, lower, spacing, balls.size(), ballsDev);

   CUDA_CHECK(hipMemcpy(&(*patch)[0], patchDev, patch->size() * sizeof(float),
                         hipMemcpyDeviceToHost));
   CUDA_CHECK(hipFree(patchDev));
   CUDA_CHECK(hipFree(ballsDev));

#if 0
   for (std::size_t i = 0; i != patch->size(); ++i) {
      assert((*patch)[i] == 0);
   }
#else
   std::cout << *patch << '\n';
#endif
}
#endif


// Note: Storing ballsSquared in constant memory does not significantly affect
// performance.
__device__
void
powerDistanceKernel(float* patch,
                    const float3 lowerCorner,
                    const float spacing,
                    const float4* ballsSquared,
                    unsigned numInfluencingBalls,
                    const unsigned* ballIndices) {
   // Calculate the Cartesian location of the grid point.
   const float3 p = {lowerCorner.x + spacing * threadIdx.x,
                     lowerCorner.y + spacing * threadIdx.y,
                     lowerCorner.z + spacing * threadIdx.z};
   // Initialize the grid value to infinity.
   float g = 1./0;
   // Find the minimum power distance over the set of balls.
   float d;
#if 0
   float4 b;
   for (unsigned i = 0; i != numInfluencingBalls; ++i) {
      b = ballsSquared[ballIndices[i]];
      d = (p.x - b.x) * (p.x - b.x) + (p.y - b.y) * (p.y - b.y) +
         (p.z - b.z) * (p.z - b.z) - b.w;
      if (d < g) {
         g = d;
      }
   }
   // Record the patch value. Convert the multi-index to a single offset.
   patch[threadIdx.x + threadIdx.y * blockDim.x +
         threadIdx.z * blockDim.x * blockDim.y] = g;
#else
   // Using shared memory improves performance on the 9600M GT, but not on
   // the 9400M.
   __shared__ float4 b[32];
   // Convert the multi-index to a single offset.
   const unsigned tid = threadIdx.x + threadIdx.y * blockDim.x +
      threadIdx.z * blockDim.x * blockDim.y;
   unsigned block;
   while (numInfluencingBalls != 0) {
      block = 32;
      if (numInfluencingBalls < 32) {
         block = numInfluencingBalls;
      }
      // Load the balls into shared memory.
      if (tid < block) {
         // Even though these are uncoalesced, this is an insignificant cost.
         b[tid] = ballsSquared[ballIndices[tid]];
      }
      __syncthreads();
      // Process the loaded balls.
      for (unsigned i = 0; i != block; ++i) {
         // This accounts for about a third of the execution time.
         d = (p.x - b[i].x) * (p.x - b[i].x) + (p.y - b[i].y) * (p.y - b[i].y) +
            (p.z - b[i].z) * (p.z - b[i].z) - b[i].w;
         if (d < g) {
            g = d;
         }
      }
      ballIndices += block;
      numInfluencingBalls -= block;
   }
   // Record the patch value. 
   // Note: This statement accounts for about 35% of execution time.
   patch[tid] = g;
#endif
}


// Compute the power distance in a neighborhood around the zero iso-surface.
// Note: Storing ballsSquared in constant memory does not significantly affect
// performance.
__device__
void
powerDistanceKernel(float* patch,
                    const float3 lowerCorner,
                    const float spacing,
                    const float4* ballsSquared,
                    unsigned numInfluencingBalls,
                    const unsigned* ballIndices,
                    const float threshold) {
   // Calculate the Cartesian location of the grid point.
   const float3 p = {lowerCorner.x + spacing * threadIdx.x,
                     lowerCorner.y + spacing * threadIdx.y,
                     lowerCorner.z + spacing * threadIdx.z};
   // Initialize the grid value to infinity.
   float g = 1./0;
   // Find the minimum power distance over the set of balls.
   float d;
   // Using shared memory improves performance on the 9600M GT, but not on
   // the 9400M.
   __shared__ float4 b[32];
   // Convert the multi-index to a single offset.
   const unsigned tid = threadIdx.x + threadIdx.y * blockDim.x +
      threadIdx.z * blockDim.x * blockDim.y;
   unsigned block;
   while (numInfluencingBalls != 0) {
      block = 32;
      if (numInfluencingBalls < 32) {
         block = numInfluencingBalls;
      }
      // Load the balls into shared memory.
      if (tid < block) {
         // Even though these are uncoalesced, this is an insignificant cost.
         b[tid] = ballsSquared[ballIndices[tid]];
      }
      __syncthreads();
      //
      // Process the loaded balls.
      //
      // Compute the distance for the first ball in the block.
      d = (p.x - b[0].x) * (p.x - b[0].x) +
         (p.y - b[0].y) * (p.y - b[0].y) +
         (p.z - b[0].z) * (p.z - b[0].z) - b[0].w;
      if (d < g) {
         g = d;
      }
      // If the distance is negative and more than the threshold from 
      // the surface, we can stop computing distances for this point.
      if (g >= -threshold) {
         // Compute the distance for the rest of the block.
         for (unsigned i = 1; i != block; ++i) {
            // This accounts for about a third of the execution time.
            d = (p.x - b[i].x) * (p.x - b[i].x) +
               (p.y - b[i].y) * (p.y - b[i].y) +
               (p.z - b[i].z) * (p.z - b[i].z) - b[i].w;
            if (d < g) {
               g = d;
            }
            // Note: Breaking out of the loop when the distance becomes less
            // than -threshold hurts performance due to loop divergence.
         }
      }

      ballIndices += block;
      numInfluencingBalls -= block;
   }
   // Record the patch value. 
   // Note: This statement accounts for about 35% of execution time.
   patch[tid] = g;
}


// Select a single patch using the block index. Then call a kernel for that
// patch.
__global__
void
powerDistanceKernel(const unsigned numRefined,
                    float* patches,
                    const uint3* indices,
                    const float3 lowerCorner,
                    const float spacing,
                    const float4* ballsSquared,
                    const unsigned* ballIndexOffsets,
                    const unsigned* packedBallIndices) {
   // Convert the 2-D block index into a single patch index.
   const unsigned i = blockIdx.x + blockIdx.y * gridDim.x;
   if (i >= numRefined) {
      return;
   }
   const unsigned begin = ballIndexOffsets[i];
   const unsigned end = ballIndexOffsets[i+1];
   const unsigned NumThreads = PatchExtent * PatchExtent * PatchExtent;
   const float3 patchLowerCorner =
      {lowerCorner.x + PatchExtent * indices[i].x * spacing,
       lowerCorner.y + PatchExtent * indices[i].y * spacing,
       lowerCorner.z + PatchExtent * indices[i].z * spacing};
   powerDistanceKernel(patches + i * NumThreads,
                       patchLowerCorner, spacing, ballsSquared,
                       end - begin, &packedBallIndices[begin]);
}


// Select a single patch using the block index. Then call a kernel for that
// patch. Compute the power distance in a neighborhood of the zero 
// iso-surface.
__global__
void
powerDistanceKernel(const unsigned numRefined,
                    float* patches,
                    const uint3* indices,
                    const float3 lowerCorner,
                    const float spacing,
                    const float4* ballsSquared,
                    const unsigned* ballIndexOffsets,
                    const unsigned* packedBallIndices,
                    const float threshold) {
   // Convert the 2-D block index into a single patch index.
   const unsigned i = blockIdx.x + blockIdx.y * gridDim.x;
   if (i >= numRefined) {
      return;
   }
   const unsigned begin = ballIndexOffsets[i];
   const unsigned end = ballIndexOffsets[i+1];
   const unsigned NumThreads = PatchExtent * PatchExtent * PatchExtent;
   const float3 patchLowerCorner =
      {lowerCorner.x + PatchExtent * indices[i].x * spacing,
       lowerCorner.y + PatchExtent * indices[i].y * spacing,
       lowerCorner.z + PatchExtent * indices[i].z * spacing};
   powerDistanceKernel(patches + i * NumThreads,
                       patchLowerCorner, spacing, ballsSquared,
                       end - begin, &packedBallIndices[begin], threshold);
}


#ifndef __CUDA_ARCH__
// Construct a level set for the power distance to a set of balls.
// The dependencies have been computed and the grid has already been refined.
void
negativePowerDistanceCuda
(Grid<float, 3, PatchExtent>* grid,
 const std::vector<geom::Ball<float, 3> >& balls,
 const container::StaticArrayOfArrays<unsigned>& dependencies,
 const float maxDistance) {
   // Dispense with the trivial case.
   if (grid->empty()) {
      return;
   }

   const std::size_t numRefined = grid->numRefined();

   // Allocate device memory for the refined patches and their indices.
   float* patchesDev;
   uint3* indicesDev;
   allocateGridCuda(*grid, numRefined, &patchesDev, &indicesDev);

   // Allocate device memory for the balls and copy the memory.
   float4* ballsSquaredDev;
   CUDA_CHECK(hipMalloc((void**)&ballsSquaredDev,
                         balls.size() * sizeof(float4)));
   {
      std::vector<float4> buffer(balls.size());
      for (std::size_t i = 0; i != balls.size(); ++i) {
         buffer[i].x = balls[i].center[0];
         buffer[i].y = balls[i].center[1];
         buffer[i].z = balls[i].center[2];
         buffer[i].w = balls[i].radius * balls[i].radius;
      }
      CUDA_CHECK(hipMemcpy(ballsSquaredDev, &buffer[0],
                            buffer.size() * sizeof(float4),
                            hipMemcpyHostToDevice));
   }

   // Allocate device memory for the ball index offsets and packed ball indices.
   // Copy the data to the device.
   unsigned* ballIndexOffsetsDev;
   unsigned* packedBallIndicesDev;
   allocateBallIndicesCuda(*grid, dependencies, &ballIndexOffsetsDev,
                           &packedBallIndicesDev);
   
   const float3 lowerCorner = {grid->lowerCorner[0], grid->lowerCorner[1],
                               grid->lowerCorner[2]};
   // Use a 2-D grid of blocks. Because the number of refined patches may 
   // exceed the maximum allowed single grid dimension.
   hipDeviceProp_t prop;
   CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
   const std::size_t Len = prop.maxGridSize[0];
   const dim3 GridDim(std::min(numRefined, Len), (numRefined + Len - 1) / Len);
   // A thread for each patch grid point.
   const dim3 ThreadsPerBlock(PatchExtent, PatchExtent, PatchExtent);

   // If we are only computing the distance in a neighborhood around the
   // zero iso-surface.
   if (maxDistance != std::numeric_limits<float>::infinity()) {
      // In computing the power distance, we only need it to be correct up to 
      // seedThreshold away from the surface. Let r be the radius of a ball,
      // d the distance from its center, and t the threshold. Suppose the
      // distance to the ball is negative. If r - d > t, then the point is
      // far inside the ball. Note however, that we are dealing with the 
      // power distance; we compute d^2 - r^2. We Manipulate the 
      // inequality.
      // r - d > t
      // d - r < -t
      // d^2 - r^2 < -t (d + r)
      // For negative distances d < r, so we use the following test.
      // d^2 - r^2 < -2 r t
      float maxRadius = 0;
      for (std::size_t i = 0; i != balls.size(); ++i) {
         if (balls[i].radius > maxRadius) {
            maxRadius = balls[i].radius;
         }
      }
      const float powerDistanceThreshold = 2 * maxRadius * maxDistance;
      // Launch the kernel for computing the distance in a neighborhood.
      powerDistanceKernel<<<GridDim,ThreadsPerBlock>>>
         (numRefined, patchesDev, indicesDev, lowerCorner, grid->spacing,
          ballsSquaredDev, ballIndexOffsetsDev, packedBallIndicesDev,
          powerDistanceThreshold);
   }
   else {
      // Launch the kernel for computing the distance for all grid points.
      powerDistanceKernel<<<GridDim,ThreadsPerBlock>>>
         (numRefined, patchesDev, indicesDev, lowerCorner, grid->spacing,
          ballsSquaredDev, ballIndexOffsetsDev, packedBallIndicesDev);
   }

   // Copy the patch data back to the host.
   CUDA_CHECK(hipMemcpy(grid->data(), patchesDev,
                         grid->numVertices() * sizeof(float),
                         hipMemcpyDeviceToHost));

   // Free the device memory.
   CUDA_CHECK(hipFree(patchesDev));
   CUDA_CHECK(hipFree(indicesDev));
   CUDA_CHECK(hipFree(ballsSquaredDev));
   CUDA_CHECK(hipFree(ballIndexOffsetsDev));
   CUDA_CHECK(hipFree(packedBallIndicesDev));
}
#endif


#ifndef __CUDA_ARCH__
// Construct a level set for the power distance to a set of balls.
void
negativePowerDistanceCuda(Grid<float, 3, PatchExtent>* grid,
                          const std::vector<geom::Ball<float, 3> >& balls) {
   // Dispense with the trivial case.
   if (grid->empty()) {
      return;
   }

   // Determine the patch/ball dependencies.
   container::StaticArrayOfArrays<unsigned> dependencies;
   {
      // 1.1 * (diagonal length of a voxel)
      const float offset = 1.1 * grid->spacing * std::sqrt(float(3));
      // Offset the ball radii to include the volume of calculated distance.
      std::vector<geom::Ball<float, 3> > offsetBalls(balls);
      for (std::size_t i = 0; i != offsetBalls.size(); ++i) {
         offsetBalls[i].radius += offset;
      }
      // Calculate the dependencies.
      patchDependencies(*grid, offsetBalls.begin(), offsetBalls.end(),
                        &dependencies);
   }

   // Refine the appropriate patches and set the rest to have an unknown
   // distance.
   grid->refine(dependencies);

   // Compute the negative power distance using the refined grid and the 
   // dependencies.
   negativePowerDistanceCuda(grid, balls, dependencies);
}
#endif


} // namespace levelSet
}
